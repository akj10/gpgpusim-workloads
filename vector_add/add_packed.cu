#include "hip/hip_runtime.h"
#include <stdio.h>
unsigned int N = 1 << 12;
unsigned int N_p = N/4;

__global__
void add(unsigned int n, unsigned int *x, unsigned int *y)
{
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = x[i] + y[i];
}

int main(void)
{
  unsigned int /**x, *y,*/ *d_x, *d_y;
  int8_t *x, *y;

  x = (int8_t*)malloc(N*sizeof(int8_t));
  y = (int8_t*)malloc(N*sizeof(int8_t));

  hipMalloc(&d_x, N_p*sizeof(unsigned int)); 
  hipMalloc(&d_y, N_p*sizeof(unsigned int));

  for (unsigned int i = 0; i < N; i++) {
    x[i] = i%256;
    y[i] = i%256;
  }

  hipMemcpy(d_x, (unsigned int*) x, N_p*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, (unsigned int*) y, N_p*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // Perform SAXPY on 1M elements
  add<<<(N_p+255)/256, 256>>>(N_p, d_x, d_y);

  hipDeviceSynchronize();
  hipMemcpy(y, d_y, N*sizeof(int8_t), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  y = (int8_t*) y;

  int8_t maxError = 0;
  for (unsigned int i = 0; i < N; i++) {
    maxError = max(maxError, (y[i]-(int8_t)(((i%256+i%256)%256))));
    if (y[i] != (int8_t)(2*(i%256))%256) printf("Elements at pos %d not matching: y[i]=%x, 2*i=%x\n", i, y[i], (2*(i%256))%256);
  }
  printf("Max error: %d\n", maxError);
}
