#include "hip/hip_runtime.h"
#include <stdio.h>

#define N (1<<16)
#define threadPerBlock (1<<8)
#define blockPerGrid min(1<<4 , (N+threadPerBlock-1) / threadPerBlock )

double cpudot(int n, float *x, float *y)
{
  double z = 0.0f;
  for (int i=0; i<n; i++) z += (double)x[i] * (double)y[i];
  return z;
}

__global__
void dot(int n, float *x, float *y, float *z)
{
  __shared__ float cache[threadPerBlock];
  unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
  cache[threadIdx.x] = 0.0;
  while (tid < n) {
    cache[threadIdx.x] += x[tid] * y[tid];
    tid += threadPerBlock * blockPerGrid;
  }
  __syncthreads();
  //printf("Thread %d: x=%f, y=%f, cache=%f\n", tid, x[tid], y[tid], cache[threadIdx.x]);

  int i = threadPerBlock/2;
  while (i != 0) {
    if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
    __syncthreads();
    //printf("iter %d: tid=%d, cache=%f\n", i, threadIdx.x, cache[threadIdx.x]);
    i /= 2;
  }
  if (threadIdx.x == 0) z[blockIdx.x] = cache[0];
}

int main(void)
{
  float *x, *y, *z, *d_x, *d_y, *d_z;
  double cpu_result, gpu_result=0.0;
  double cpu_acc=0.0;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  z = (float*)malloc(blockPerGrid*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_z, blockPerGrid*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 0.5;
    y[i] = i;
    z[i%blockPerGrid] = 0;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, blockPerGrid*sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // Perform dot on 1M elements
  dot<<<blockPerGrid, threadPerBlock>>>(N, d_x, d_y, d_z);

  hipDeviceSynchronize();
  hipMemcpy(z, d_z, blockPerGrid*sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i=0; i<blockPerGrid; i++) {
    gpu_result += (double)z[i];
  }

  cpu_result = cpudot(N, x, y);
    if (abs(gpu_result - cpu_result) > 0.000001) printf("GPU Dot product:%f not matching with CPU:%f\n", gpu_result, cpu_result);
    else printf("GPU Dot product:%f matches with CPU:%f\n", gpu_result, cpu_result);
}
