#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 16
#define threadPerBlock 16
#define blockPerGrid min(N , (N+threadPerBlock-1) / threadPerBlock )

float cpudot(int n, float *x, float *y)
{
  float z = 0.0f;
  for (int i=0; i<n; i++) z += x[i] * y[i];
  return z;
}

__global__
void dot(int n, float *x, float *y, float *z)
{
  __shared__ float cache[threadPerBlock];
  unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (tid < n) cache[threadIdx.x] = x[tid] * y[tid];
  __syncthreads();
  //printf("Thread %d: x=%f, y=%f, cache=%f\n", tid, x[tid], y[tid], cache[threadIdx.x]);

  int i = threadPerBlock/2;
  while (i != 0) {
    if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
    __syncthreads();
    //printf("iter %d: tid=%d, cache=%f\n", i, threadIdx.x, cache[threadIdx.x]);
    i /= 2;
  }
  if (threadIdx.x == 0) z[blockIdx.x] = cache[0];
}

int main(void)
{
  float *x, *y, *z, *d_x, *d_y, *d_z;
  float cpu_result, gpu_result;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  z = (float*)malloc(blockPerGrid*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_z, blockPerGrid*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 0.5;
    y[i] = i;
    z[i%blockPerGrid] = 0;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, blockPerGrid*sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // Perform dot on 1M elements
  dot<<<blockPerGrid, threadPerBlock>>>(N, d_x, d_y, d_z);

  hipDeviceSynchronize();
  hipMemcpy(z, d_z, blockPerGrid*sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i=0; i<blockPerGrid; i++) printf("%f\n", z[i]);//gpu_result += z[i];

  cpu_result = cpudot(N, x, y);
    if (gpu_result - cpu_result > 1e-6) printf("GPU Dot product:%f not matching with CPU:%f\n", *z, cpu_result);
    else printf("GPU Dot product matches with CPU: %f\n");
}
